
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;

__global__ void hello_from_gpu()
{
    const int b = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
   // cout<<b<<endl;
    printf("Hello World from block-%d and thread-(%d, %d)!\n", b, tx, ty);
}

int main(void)
{
    const dim3 block_size(2, 4);
    hello_from_gpu<<<2, block_size>>>();
    hipDeviceSynchronize();
    return 0;
}
